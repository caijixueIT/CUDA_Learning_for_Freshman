#include <bits/stdc++.h>
#include <hip/hip_runtime.h>


#include <time.h>
#include <sys/time.h>


namespace random_utils {
std::random_device rd;
std::mt19937 gen(rd());
// 定义浮点数分布范围（例如 0.0 到 1.0）
std::uniform_real_distribution<> dis(-10.f, 10.f);

float rand_float() {
    return dis(gen);
}
};


// grid(M / BM, N / BN)
// block(BM, BN)
// A [M, K]
// B [K, N]
// C [M, N]
template<int BM, int BN, int BK>
__global__ void gemm(float* A, float* B, float* C, int M, int N, int K) {
    __shared__ float sA[BM][BK];
    __shared__ float sB[BK][BN];

    int tx = threadIdx.x / BN;
    int ty = threadIdx.x % BN;

    int A_row = blockIdx.x * BM + tx;
    int B_col = blockIdx.y * BN + ty;

    float val = 0.f;
    for (int k = 0; k < K / BK; ++k) {
        for (int i = ty; i < BK; i += BN) {
            int A_col = k * BK + i;
            sA[tx][i] = A[A_row * K + A_col];
        }
        for (int i = tx; i < BK; i += BM) {
            int B_row = k * BK + i;
            sB[i][ty] = B[B_row * N + B_col];
        }
        __syncthreads();

        for (int i = 0; i < BK; ++i) {
            val += sA[tx][i] * sB[i][ty];
        }
        __syncthreads();
    }

    C[A_row * N + B_col] = val;
}

void gemm_cpu(float* A, float* B, float* C, int M, int N, int K) {
    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            float val = 0.f;
            for (int k = 0; k < K; ++k) {
                val += A[i * K + k] * B[k * N + j];
            }
            C[i * N + j] = val;
        }
    }
}

void init_matric(float* A, int row, int col) {
    for (int i = 0; i < row; ++i) {
        for (int j = 0; j < col; ++j) {
            A[i * col + j] = random_utils::rand_float();
        }
    }
}

bool check(float* res_gpu, float* res_cpu, int M, int N) {
    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            // printf("(%d, %d) %f %f\n", i, j, res_gpu[i * N + j], res_cpu[i * N + j]);
            if (fabs(res_gpu[i * N + j] - res_cpu[i * N + j]) > 2e-3) {
                printf("(%d, %d) %f %f\n", i, j, res_gpu[i * N + j], res_cpu[i * N + j]);
                return false;
            }
        }
    }
    return true;
}

void print_matric(float* A, int row, int col) {
    for (int i = 0; i < row; ++i) {
        for (int j = 0; j < col; ++j) {
            printf("%f ", A[i * col + j]);
        }
        printf("\n");
    }
    printf("####################\n");
}

int main() {
    int M = 1024 * 2;
    int N = 1024 * 2;
    int K = 1024 * 1;
    constexpr int BM = 32;
    constexpr int BN = 32;
    constexpr int BK = 8;

    // allocate memory in cpu
    float* A = (float*)malloc(M * K * sizeof(float));
    float* B = (float*)malloc(K * N * sizeof(float));
    float* C = (float*)malloc(M * N * sizeof(float));
    float* C_ref = (float*)malloc(M * N * sizeof(float));

    // init cpu data
    init_matric(A, M, K);
    init_matric(B, K, N);
    // print_matric(A, M, K);
    // print_matric(B, K, N);

    // allocate memory in gpu
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, M * K * sizeof(float));
    hipMalloc((void**)&d_B, K * N * sizeof(float));
    hipMalloc((void**)&d_C, M * N * sizeof(float));

    // init gpu data
    hipMemcpy(d_A, A, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, K * N * sizeof(float), hipMemcpyHostToDevice);

    // gemm by gpu
    dim3 grid(M/BM, N/BN);
    dim3 block(BM * BN);
    gemm<BM, BN, BK><<<grid, block>>>(d_A, d_B, d_C, M, N, K);
    hipDeviceSynchronize();
    hipMemcpy(C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost);
    // print_matric(C, M, N);

    // gemm by cpu
    gemm_cpu(A, B, C_ref, M, N, K);
    // print_matric(C_ref, M, N);

    // check
    if (check(C, C_ref, M, N)) {
        printf("pass\n");
    } else {
        printf("fail\n");
    }

    // profile
    int TEST_TIMES = 100;
    for (int i = 0; i < TEST_TIMES; ++i) {
        gemm<BM, BN, BK><<<grid, block>>>(d_A, d_B, d_C, M, N, K);
    }
    float time_elapsed=0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0); // 记录开始时间
    for (int i = 0; i < TEST_TIMES; ++i) {
        gemm<BM, BN, BK><<<grid, block>>>(d_A, d_B, d_C, M, N, K);
    }
    hipEventRecord(stop, 0); // 记录结束时间
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_elapsed, start, stop); // 计算时间差
    std::cout << "elasped time = " << time_elapsed / TEST_TIMES << "ms" << std::endl;

    return 0;
}