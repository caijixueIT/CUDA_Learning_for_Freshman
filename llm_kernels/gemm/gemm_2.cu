#include <bits/stdc++.h>
#include <hip/hip_runtime.h>


#include <time.h>
#include <sys/time.h>


namespace random_utils {
std::random_device rd;
std::mt19937 gen(rd());
// 定义浮点数分布范围（例如 0.0 到 1.0）
std::uniform_real_distribution<> dis(-10.f, 10.f);

float rand_float() {
    return dis(gen);
}
};


// 每个 block 计算 C 中大小为 BM * BN 的子矩阵, block 个数 = (M / BM) * (N / BN)
// 每个 thread 计算 C 中大小为 TM * TN 的子矩阵, thread 个数 = (BM / TM) * (BN / TN)
// grid(M / BM, N / BN)
// block(BM / TM, BN / TN)
// A [M, K]
// B [K, N]
// C [M, N]
template<int BM=32, int BN=32, int BK=4, int TM=4, int TN=4>
__global__ void gemm(float* A, float* B, float* C, int M, int N, int K) {
    __shared__ float sA[BM][BK];
    __shared__ float sB[BK][BN];
    
    float vals[TM][TN] = {0.f};
    for (int k = 0; k < K / BK; ++k) {
        for (int i = threadIdx.x; i < BM; i += TM) {
            for (int j = threadIdx.y; j < BN; j += TN) {
                int A_row = blockIdx.x * BM + i;
                int B_col = blockIdx.y * BN + j;

                for (int l = threadIdx.y; l < BK; l +=TN) {
                    int A_col = k * BK + l;
                    sA[i][l] = A[A_row * K + A_col];
                }

                for (int l = threadIdx.x; l < BK; l += TM) {
                    int B_row = k * BK + l;
                    sB[l][j] = B[B_row * N + B_col];
                }
            }
        }
        __syncthreads();
        
        for (int i = 0; i < TM; ++i) {
            for (int j = 0; j < TN; ++j) {
                for (int l = 0; l < BK; l++) {
                    vals[i][j] += sA[i * BM / TM + threadIdx.x][l] * sB[l][j * BN / TN + threadIdx.y];
                }
            }
        }
        __syncthreads();
    }

    for (int i = 0; i < TM; ++i) {
        for (int j = 0; j < TN; ++j) {
            int C_row = blockIdx.x * BM + i * (BM / TM) + threadIdx.x;
            int C_col = blockIdx.y * BN + j * (BN / TN) + threadIdx.y;
            C[C_row * N + C_col] = vals[i][j];
        }
    }
}

void gemm_cpu(float* A, float* B, float* C, int M, int N, int K) {
    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            float val = 0.f;
            for (int k = 0; k < K; ++k) {
                val += A[i * K + k] * B[k * N + j];
            }
            C[i * N + j] = val;
        }
    }
}

void init_matric(float* A, int row, int col) {
    for (int i = 0; i < row; ++i) {
        for (int j = 0; j < col; ++j) {
            A[i * col + j] = random_utils::rand_float();
        }
    }
}

void init_identity_matric(float* A, int row, int col) {
    for (int i = 0; i < row; ++i) {
        for (int j = 0; j < col; ++j) {
            A[i * col + j] = (i == j ? 1.f : 0.f);
        }
    }
}

bool check(float* res_gpu, float* res_cpu, int M, int N) {
    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            if (fabs(res_gpu[i * N + j] - res_cpu[i * N + j]) > 1e-2) {
                printf("(%d, %d) %f %f\n", i, j, res_gpu[i * N + j], res_cpu[i * N + j]);
                return false;
            }
        }
    }
    return true;
}

void print_matric(float* A, int row, int col) {
    return;
    // for (int i = 0; i < row; ++i) {
    //     for (int j = 0; j < col; ++j) {
    //         printf("%f ", A[i * col + j]);
    //     }
    //     printf("\n");
    // }
}

int main() {
    int M = 1024 * 2;
    int N = 1024 * 2;
    int K = 1024 * 1;
    constexpr int BM = 32;
    constexpr int BN = 32;
    constexpr int BK = 8;
    constexpr int TM = 4;
    constexpr int TN = 4;

    // allocate memory in cpu
    float* A = (float*)malloc(M * K * sizeof(float));
    float* B = (float*)malloc(K * N * sizeof(float));
    float* C = (float*)malloc(M * N * sizeof(float));
    float* C_ref = (float*)malloc(M * N * sizeof(float));

    // init cpu data
    init_matric(A, M, K);
    init_matric(B, K, N);
    print_matric(A, M, K);
    print_matric(B, K, N);

    // allocate memory in gpu
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, M * K * sizeof(float));
    hipMalloc((void**)&d_B, K * N * sizeof(float));
    hipMalloc((void**)&d_C, M * N * sizeof(float));

    // init gpu data
    hipMemcpy(d_A, A, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, K * N * sizeof(float), hipMemcpyHostToDevice);

    // gemm by gpu
    dim3 grid(M/BM, N/BN);
    dim3 block(BM/TM, BN/TN);
    gemm<BM, BN, BK, TM, TN><<<grid, block>>>(d_A, d_B, d_C, M, N, K);
    hipDeviceSynchronize();
    hipMemcpy(C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost);
    print_matric(C, M, N);

    // gemm by cpu
    gemm_cpu(A, B, C_ref, M, N, K);
    print_matric(C_ref, M, N);

    // check
    if (check(C, C_ref, M, N)) {
        printf("pass\n");
    } else {
        printf("fail\n");
    }

    // profile
    int TEST_TIMES = 100;
    for (int i = 0; i < TEST_TIMES; ++i) {
        gemm<BM, BN, BK, TM, TN><<<grid, block>>>(d_A, d_B, d_C, M, N, K);
    }
    float time_elapsed=0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0); // 记录开始时间
    for (int i = 0; i < TEST_TIMES; ++i) {
        gemm<BM, BN, BK, TM, TN><<<grid, block>>>(d_A, d_B, d_C, M, N, K);
    }
    hipEventRecord(stop, 0); // 记录结束时间
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_elapsed, start, stop); // 计算时间差
    std::cout << "elasped time = " << time_elapsed / TEST_TIMES << "ms" << std::endl;

    return 0;
}