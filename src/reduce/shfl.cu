#include <bits/stdc++.h>
#include <hip/hip_runtime.h>


#include <time.h>
#include <sys/time.h>

__global__ void shfl(float* arr, int n) {
    int tid = threadIdx.x;
    float val = tid < n ? arr[tid] : 0.f;
    val = __shfl_down_sync(0xffffffff, val, 4, 16);
    arr[tid] = val;
}

int main() {
    constexpr int N = 32;
    float* arr = (float*)malloc(sizeof(float) * N);
    for (int i = 0; i < N; i++) {
        arr[i] = i + 1;
    }

    for (int i = 0; i < 4; i++) {
        for (int j = 0; j < 8; j++) {
            printf("%.2f  ", arr[i * 8 + j]);
        }
        printf("\n");
    }
    printf("########################################\n");
    float* d_arr;
    hipMalloc((void**)&d_arr, sizeof(float) * N);
    hipMemcpy(d_arr, arr, sizeof(float) * N, hipMemcpyHostToDevice);
    shfl<<<1, N>>>(d_arr, N);
    hipDeviceSynchronize();
    hipMemcpy(arr, d_arr, sizeof(float) * N, hipMemcpyDeviceToHost);

    for (int i = 0; i < 4; i++) {
        for (int j = 0; j < 8; j++) {
            printf("%.2f  ", arr[i * 8 + j]);
        }
        printf("\n");
    }
    
    return 0;
}