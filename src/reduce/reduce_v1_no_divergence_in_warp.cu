#include <bits/stdc++.h>
#include <hip/hip_runtime.h>


#include <time.h>
#include <sys/time.h>

#define THREAD_PER_BLOCK 256

__global__ void reduce_sum_kernel(float* arr, int n, float* res) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    
    __shared__ float smem[THREAD_PER_BLOCK];

    smem[tid] = arr[idx];
    __syncthreads();

    for (int i = 1; i < THREAD_PER_BLOCK; i = i * 2) {
        int x = tid * i * 2;
        if (x < THREAD_PER_BLOCK) {
            smem[x] += smem[x + i];
        }
        __syncthreads();
    }

    if (tid == 0) {
        res[blockIdx.x] = smem[0];
    }
}

bool check(float *out, float *res, int N){
    for(int i=0; i<N; i++){
        // printf("out[%d]=%f, ref[%d]=%f\n", i, out[i], i, res[i]);
        if(out[i]!= res[i])
            return false;
    }
    return true;
}

int main() {
    constexpr int N = 1024 * 1024;
    constexpr int threads_per_block = THREAD_PER_BLOCK;

    float* arr = (float*)malloc(N * sizeof(float));
    for (int i = 0; i < N; ++i) {
        arr[i] = 1; // i * (N - 1) / (float) (N * 10);
    }
    
    float* d_arr;
    hipMalloc((void**)&d_arr, N * sizeof(float));
    hipMemcpy(d_arr, arr, N * sizeof(float), hipMemcpyHostToDevice);

    float* out = (float*)malloc(N / threads_per_block * sizeof(float));
    float* out_ref = (float*)malloc(N / threads_per_block * sizeof(float));

    float* d_out;
    hipMalloc((void**)&d_out, N / threads_per_block * sizeof(float));

    for (int i = 0; i < N / threads_per_block; ++i) {
        float sum = 0.f;
        for (int j = 0; j < threads_per_block; ++j) {
            sum += arr[i * threads_per_block + j];
        }
        out_ref[i] = sum;
    }

    dim3 grid_dim(N / threads_per_block);
    dim3 block_dim(threads_per_block);

    reduce_sum_kernel<<<grid_dim, block_dim>>>(d_arr, N, d_out);
    hipMemcpy(out, d_out, N/threads_per_block * sizeof(float), hipMemcpyDeviceToHost);

    if (check(out, out_ref, N/threads_per_block)) {
        std::cout << "succeed!" << std::endl;
    } else {
        std::cout << "failed!" << std::endl;
    }

    int TEST_TIMES = 100;
    for (int i = 0; i < TEST_TIMES; ++i) {
        reduce_sum_kernel<<<grid_dim, block_dim>>>(d_arr, N, d_out);
    }
    float time_elapsed=0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start,0);    //记录当前时间
    for (int i = 0; i < TEST_TIMES; ++i) {
        reduce_sum_kernel<<<grid_dim, block_dim>>>(d_arr, N, d_out);
    }
    hipEventRecord(stop,0);    //记录当前时间
    hipEventSynchronize(start);    //Waits for an event to complete.
    hipEventSynchronize(stop);     //Waits for an event to complete.Record之前的任务
    hipEventElapsedTime(&time_elapsed, start, stop);    //计算时间差
    std::cout << "reduce_sum_kernel elasped time = " << time_elapsed/TEST_TIMES << "ms" << std::endl;

    hipFree(d_arr);
    hipFree(d_out);
    free(arr);
    free(out);
    free(out_ref);
    return 0;
}