#include <bits/stdc++.h>
#include <hip/hip_runtime.h>


#include <time.h>
#include <sys/time.h>

#define WARPS_PER_BLOCK 4
#define THREADS_PER_BLOCK (WARPS_PER_BLOCK * 32)

template <typename T, unsigned int WarpSize>
__device__ __forceinline__ T warpReduceSum(T sum) {
    if (WarpSize >= 32) sum += __shfl_down_sync(0xffffffff, sum, 16); // 0-16, 1-17, 2-18, etc.
    if (WarpSize >= 16) sum += __shfl_down_sync(0xffffffff, sum, 8);// 0-8, 1-9, 2-10, etc.
    if (WarpSize >= 8) sum += __shfl_down_sync(0xffffffff, sum, 4);// 0-4, 1-5, 2-6, etc.
    if (WarpSize >= 4) sum += __shfl_down_sync(0xffffffff, sum, 2);// 0-2, 1-3, 4-6, 5-7, etc.
    if (WarpSize >= 2) sum += __shfl_down_sync(0xffffffff, sum, 1);// 0-1, 2-3, 4-5, etc.
    return sum;
}

/*
    A : M * N
    x : N * 1
    y : M * 1
    一个 warp 处理多行
*/
template <int ROWS_PER_WARP = 4>
__global__ void gemv_warp(float *A, float *x, float *y, int M, int N) {
    int tid = threadIdx.x;
    int laneId = tid % 32;
    int warpId = tid / 32;

    constexpr int kWarpSize = 32 / ROWS_PER_WARP; // 把一个线程 warp 分成更小的 group
    int kLaneId = laneId % kWarpSize;  // group 内的线程编号
    int kWarpId = laneId / kWarpSize;  // group id

    int row = blockIdx.x * WARPS_PER_BLOCK * ROWS_PER_WARP + warpId * ROWS_PER_WARP + kWarpId;
    if (row >= M) return;

    int col = kLaneId * 4;
    float sum = 0;
    for (int i = col; i < N; i += kWarpSize * 4) {
        float4 a = *(float4*)(A + row * N + i);
        float4 b = *(float4*)(x + i);
        sum += a.x * b.x + a.y * b.y + a.z * b.z + a.w * b.w;
    }
    sum = warpReduceSum<float, kWarpSize>(sum);
    if (kLaneId == 0) y[row] = sum;
}

void gemv_cpu(float *A, float *x, float *y, int M, int N) {
    for (int i = 0; i < M; i++) {
        float sum = 0;
        for (int j = 0; j < N; j++) {
            sum += A[i * N + j] * x[j];
        }
        y[i] = sum;
    }
}

int check(float* y, float *y_ref, int M) {
    for (int i = 0; i < M; i++) {
        if (fabs(y_ref[i] - y[i]) > 1e-5) {
            printf("error at %d: %f %f\n", i, y_ref[i], y[i]);
            return 0;
        }
    }
    return 1;
}

int main() {
    int M = 4096;
    int N = 1024;
    float *A = (float*)malloc(M * N * sizeof(float));
    float *x = (float*)malloc(N * sizeof(float));
    float *y = (float*)malloc(M * sizeof(float));
    float *y_ref = (float*)malloc(M * sizeof(float));

    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            A[i * N + j] = rand() % 11;
        }
    }

    for (int i = 0; i < N; i++) {
        x[i] = rand() % 13;
    }

    gemv_cpu(A, x, y_ref, M, N);

    float *d_A, *d_x, *d_y;
    hipMalloc(&d_A, M * N * sizeof(float));
    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, M * sizeof(float));
    hipMemcpy(d_A, A, M * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);

    constexpr int ROWS_PER_WARP = 4;
    constexpr int ROWS_PER_BLOCK = ROWS_PER_WARP * WARPS_PER_BLOCK;

    const int blocks = (M + ROWS_PER_BLOCK - 1) / ROWS_PER_BLOCK;
    gemv_warp<ROWS_PER_WARP><<<blocks, THREADS_PER_BLOCK>>>(d_A, d_x, d_y, M, N);
    hipMemcpy(y, d_y, M * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    if (check(y, y_ref, M)) printf("passed\n");
    else printf("failed\n");

    int TEST_TIMES = 100;
    for (int i = 0; i < TEST_TIMES; ++i) {
        gemv_warp<ROWS_PER_WARP><<<blocks, THREADS_PER_BLOCK>>>(d_A, d_x, d_y, M, N);
    }
    float time_elapsed=0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start,0);                               // 记录开始时间
    for (int i = 0; i < TEST_TIMES; ++i) {
        gemv_warp<ROWS_PER_WARP><<<blocks, THREADS_PER_BLOCK>>>(d_A, d_x, d_y, M, N);
    }
    hipEventRecord(stop,0);                                // 记录结束时间
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_elapsed, start, stop);       // 计算时间差
    std::cout << "elasped time = " << time_elapsed/TEST_TIMES << "ms" << std::endl;

    return 0;
}