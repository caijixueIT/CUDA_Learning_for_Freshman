#include <bits/stdc++.h>
#include <hip/hip_runtime.h>


#include <time.h>
#include <sys/time.h>
#include <hipblas.h>

void gemm_cpu(float* A, float* B, float* C, int M, int N, int K) {
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            float val = 0;
            for (int k = 0; k < K; k++) {
                val += A[i * K + k] * B[k * N + j];
            }
            C[i * N + j] = val;
        }
    }
}

int check(float* C, float* C_ref, int M, int N) {
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            // if (C[i * N + j] != C_ref[i * N + j]) {
            //     return 0;
            // }
            // if (i < 10 && j < 10) {
            //     printf("%f %f\n", C[i * N + j], C_ref[i * N + j]);
            // }
            if (fabs(C[i * N + j] - C_ref[i * N + j]) > 0.0001) {
                return 0;
            }
        }
    }
    return 1;
}

int main() {
    int M = 1024;
    int N = 1024;
    int K = 1024;
    float* A = (float*)malloc(M * K * sizeof(float));
    float* B = (float*)malloc(K * N * sizeof(float));
    float* C = (float*)malloc(M * N * sizeof(float));
    float* C_ref = (float*)malloc(M * N * sizeof(float));
    for (int i = 0; i < M * K; i++) {
        A[i] = (rand() % 17) / 23.3;
    }
    for (int i = 0; i < K * N; i++) {
        B[i] = (rand() % 23) / 17.7;
    }
    for (int i = 0; i < M * N; i++) {
        C[i] = 0;
    }
    float* d_A;
    float* d_B;
    float* d_C;
    hipMalloc((void**)&d_A, M * K * sizeof(float));
    hipMalloc((void**)&d_B, K * N * sizeof(float));
    hipMalloc((void**)&d_C, M * N * sizeof(float));
    hipMemcpy(d_A, A, M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, K * N * sizeof(float), hipMemcpyHostToDevice);

    hipError_t cudaStat;  // cudaMalloc status
    hipblasStatus_t stat;   // cuBLAS functions status
    hipblasHandle_t handle; // cuBLAS context

    float alpha = 1.0;
    float beta = 0.0;
    stat = hipblasCreate(&handle); // initialize CUBLAS context
    stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, d_B, N, d_A, K, &beta, d_C, N);
    hipMemcpy(C, d_C, M * N * sizeof(float), hipMemcpyDeviceToHost);

    gemm_cpu(A, B, C_ref, M, N, K);

    if (check(C, C_ref, M, N)) {
        printf("Correct!\n");
    } else {
        printf("Wrong!\n");
    }

    int TEST_TIMES = 100;
    for (int i = 0; i < TEST_TIMES; ++i) {
        stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, d_B, N, d_A, K, &beta, d_C, N);
    }
    float time_elapsed=0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start,0);                               // 记录开始时间
    for (int i = 0; i < TEST_TIMES; ++i) {
        stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, d_B, N, d_A, K, &beta, d_C, N);
    }
    hipEventRecord(stop,0);                                // 记录结束时间
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_elapsed, start, stop);       // 计算时间差
    std::cout << "elasped time = " << time_elapsed/TEST_TIMES << "ms" << std::endl;
    return 0;
}